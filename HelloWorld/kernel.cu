
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <windows.h>

#define ALPHA 1 // 0.8
#define BETA  1 // 0.2

void printMatrix(float ** M, int rowSize, int columnSize);
void doCPUStencil(float ** M, int rowSize, int columnSize);
float getTopElement(float ** M, int indexOfRow, int indexOfColumn, int rowSize, int columnSize);
float getRightElement(float ** M, int indexOfRow, int indexOfColumn, int rowSize, int columnSize);
float getBottomElement(float ** M, int indexOfRow, int indexOfColumn, int rowSize, int columnSize);
float getLeftElement(float ** M, int indexOfRow, int indexOfColumn, int rowSize, int columnSize);

int main()
{
	int const rowSize    = 1024;
	int const columnSize = 1024;

	float **M = (float **)malloc(sizeof(float *) * rowSize);

	for (int i = 0; i < rowSize; i++) {
		// �����ȣ�Row Major��
		M[i] = (float *)malloc(sizeof(float) * columnSize);

		for (int j = 0; j < columnSize; j++) {
			M[i][j] = (float) i * 10 + j;
		}
	}

	/*printf("Origin Matrix:\n");
	printMatrix(M, rowSize, columnSize);*/

	doCPUStencil(M, rowSize, columnSize);

	/*printf("Transformed Matrix:\n");
	printMatrix(M, rowSize, columnSize);*/
	

	for (int i = 0; i < rowSize; i++) {
		free(M[i]);
	}
	free(M);

	return 0;
}

void printMatrix(float ** M, int rowSize, int columnSize) {
	for (int i = 0; i < rowSize; i++) {
		for (int j = 0; j < columnSize; j++) {
			printf("%5.0f ", M[i][j]);
		}
		printf("\n");
	}
	printf("\n");
}

void doCPUStencil(float ** M, int rowSize, int columnSize) {
	LARGE_INTEGER m_liPerfFreq = { 0 };
	//��ȡÿ�����CPU Performance Tick
	QueryPerformanceFrequency(&m_liPerfFreq);
	LARGE_INTEGER m_liPerfStart = { 0 };
	QueryPerformanceCounter(&m_liPerfStart);

	float * tmp = (float *)malloc(sizeof(float) * columnSize);

	for (int i = 0; i < rowSize; i++) {
		for (int j = 0; j < columnSize; j++) {
            tmp[j] = ALPHA * M[i][j] + BETA * (
				getTopElement(M, i, j, rowSize, columnSize) +
				getRightElement(M, i, j, rowSize, columnSize) +
				getBottomElement(M, i, j, rowSize, columnSize) +
				getLeftElement(M, i, j, rowSize, columnSize));
		}
		for (int j = 0; j < columnSize; j++) {
			M[i][j] = tmp[j];
		}
	}

	free(tmp);

	LARGE_INTEGER liPerfNow = { 0 };
	// ����CPU���е����ڵ�ʱ��
	QueryPerformanceCounter(&liPerfNow);
	int time = (((liPerfNow.QuadPart - m_liPerfStart.QuadPart) * 1000) / m_liPerfFreq.QuadPart);
	char buffer[100];
	sprintf(buffer, "ִ��ʱ�䣺 %d millisecond.\n", time);
	printf(buffer);
}

float getTopElement(float ** M, int indexOfRow, int indexOfColumn, int rowSize, int columnSize) {
	if (indexOfRow == 0) {
		return 0.0;
	}
	else {
		return M[indexOfRow - 1][indexOfColumn];
	}
}
float getRightElement(float ** M, int indexOfRow, int indexOfColumn, int rowSize, int columnSize) {
	if (indexOfColumn == columnSize - 1) {
		return 0.0;
	}
	else {
		return M[indexOfRow][indexOfColumn + 1];
	}
}
float getBottomElement(float ** M, int indexOfRow, int indexOfColumn, int rowSize, int columnSize) {
	if (indexOfRow == rowSize - 1) {
		return 0.0;
	}
	else {
		return M[indexOfRow + 1][indexOfColumn];
	}
}
float getLeftElement(float ** M, int indexOfRow, int indexOfColumn, int rowSize, int columnSize) {
	if (indexOfColumn == 0) {
		return 0.0;
	}
	else {
		return M[indexOfRow][indexOfColumn - 1];
	}
}
