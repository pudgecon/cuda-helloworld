#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#define MATRIX_SIZE 10
#define ELEMENT_COUNT (MATRIX_SIZE * MATRIX_SIZE)

#define ALAPH 0.8
#define BETA  0.2

void doCPUStencil(float * M, int matrixSize);
void printMatrix(float * M, int matrixSize);
float getTopElement(float * M, int index, int matrixSize);
float getRightElement(float * M, int index, int matrixSize);
float getBottomElement(float * M, int index, int matrixSize);
float getLeftElement(float * M, int index, int matrixSize);
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int main()
{
	float *M = (float *)malloc(sizeof(float) * ELEMENT_COUNT);
	for (int i = 0; i < ELEMENT_COUNT; i++) {
		M[i] = (float)i;
	}

	printMatrix(M, MATRIX_SIZE);

	doCPUStencil(M, MATRIX_SIZE);

	printMatrix(M, MATRIX_SIZE);

	return 0;
}

void printMatrix(float * M, int matrixSize) {
	for (int i = 0; i < matrixSize * matrixSize; i++) {
		printf("%10.0f ", M[i]);

		if (i % matrixSize + 1 == matrixSize)
		{
			printf("\n");
		}
	}

	printf("\n");
}

void doCPUStencil(float * M, int matrixSize) {
	for (int i = 0; i < matrixSize * matrixSize; i++) {
		M[i] = ALAPH * M[i] + BETA * (
			getTopElement(M, i, matrixSize) +
			getRightElement(M, i, matrixSize) +
			getBottomElement(M, i, matrixSize) +
			getLeftElement(M, i, matrixSize));
	}
}


float getTopElement(float * M, int index, int matrixSize) {
	if (index < matrixSize) {
		return 0.0;
	}
	else {
		return M[index - matrixSize];
	}
}
float getRightElement(float * M, int index, int matrixSize) {
	if ((index + 1) % matrixSize == 0) {
		return 0.0;
	}
	else {
		return M[index + 1];
	}
}
float getBottomElement(float * M, int index, int matrixSize) {
	if (index >= matrixSize * (matrixSize - 1)) {
		return 0.0;
	}
	else {
		return M[index + matrixSize];
	}
}
float getLeftElement(float * M, int index, int matrixSize) {
	if (index % matrixSize == 0) {
		return 0.0;
	}
	else {
		return M[index - 1];
	}
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> >(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
